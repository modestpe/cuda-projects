#include "hip/hip_runtime.h"
// This program is used to get performance numbers for different
// implementations of sum reduction in CUDA
// By: Nick from CoffeeBeforeArch

#include <iostream>
#include <fstream>
#include "common.h"

int main(){
    // Number of iterations to run per-kernel (10 by default)
    int N = 10;

    // Upper bound of array size (2^20 by default)
    int D = 1 << 20;

    // Vector to get return average execution times
    vector<float> times;

    // Get execution time for naive implementation
    times = launch_perf_test(D, N);

    // Write out the times to a data file
    ofstream output_file;
    output_file.open("timing.dat", ios::out | ios::trunc);
    for(auto i : times){
        output_file << i << "\n"; 
    }

    return 0;
}
